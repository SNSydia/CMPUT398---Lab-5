#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
    { gpuAssert((ans), __FILE__, __LINE__); }


__global__ void histogram_kernel_shared(unsigned int *input, unsigned int *bins, unsigned int inputLength, unsigned int binLength)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    int stride = blockDim.x * gridDim.x;

    extern __shared__ unsigned int binsShared[]; //extern??

	for (int x = threadIdx.x; x < binLength; x += blockDim.x)
	{
		binsShared[x] = 0;
	}

	__syncthreads();

    for (int x = i; x < inputLength; x += stride)
    {
        atomicAdd(&(binsShared[input[x]]), 1);
    }

    __syncthreads();

    for (int x = threadIdx.x; x < binLength; x += blockDim.x)
    {

        atomicAdd(&(bins[x]), binsShared[x]);
    }

    __syncthreads();




}

__global__ void truncate(unsigned int *input, unsigned int *bins, unsigned int inputLength, unsigned int binLength)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < binLength)
	{
		if (bins[i] > 127)
			bins[i] = 127;
	}
}

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins, unsigned int inputLength, unsigned int binLength)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    int stride = blockDim.x * gridDim.x;

	__syncthreads();

 
	if (i < binLength)
    {
        atomicAdd(&(bins[input[i]]), 1);
	
		if (i == 0)
			bins[0] = 0;

    }

	bins[0] = 0; //remove


    __syncthreads();


    if (i < binLength)
    {
		if (bins[i] > 127)
		{
			bins[i] = 127;
		}
    }


}


inline void gpuAssert(hipError_t code, const char *file, int line,
	bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
			file, line);
		if (abort)
			exit(code);
	}
}

int main(int argc, char *argv[]) {
	wbArg_t args;
	int inputLength;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
		&inputLength, "Integer");
	hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc((void**) &deviceInput, inputLength * sizeof(unsigned int));
	hipMalloc((void**) &deviceBins, NUM_BINS * sizeof(unsigned int));

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here

    hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(deviceBins, hostBins, NUM_BINS * sizeof(unsigned int), hipMemcpyHostToDevice);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	// ----------------------------------------------------------
	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");

	// TODO: Perform kernel computation here

	hipMemset((void *)deviceBins, 0, NUM_BINS * sizeof(unsigned int));
	memset((void *)hostBins, 0, NUM_BINS * sizeof(unsigned int));
	

	CUDA_CHECK(hipDeviceSynchronize());

    dim3 gridDim(32);
    dim3 blockDim(512);

    histogram_kernel_shared<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
	truncate << <gridDim, blockDim, NUM_BINS * sizeof(unsigned int) >> >(deviceInput, deviceBins, inputLength, NUM_BINS);


	// You should call the following lines after you call the kernel.
	// CUDA_CHECK(hipGetLastError());
	// CUDA_CHECK(hipDeviceSynchronize());

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here

    hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here

    hipFree(deviceInput);
    hipFree(deviceBins);

	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
